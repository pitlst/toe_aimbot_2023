#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include "common_structs.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

using namespace cv;
using namespace std;

__device__ float calc_iou(float* a, float *b)
{
    // 获取矩形框a的坐标
    float x0_a = a[0];
    float y0_a = a[1];
    float x1_a = a[2];
    float y1_a = a[3];
    
    // 获取矩形框b的坐标
    float x0_b = b[0];
    float y0_b = b[1];
    float x1_b = b[2];
    float y1_b = b[3];
    
    // 计算交集面积
    float intersectionArea = max(0.0f, min(x1_a, x1_b) - max(x0_a, x0_b)) * max(0.0f, min(y1_a, y1_b) - max(y0_a, y0_b));
    
    // 计算并集面积
    float area_a = (x1_a - x0_a) * (y1_a - y0_a);
    float area_b = (x1_b - x0_b) * (y1_b - y0_b);
    float unionArea = area_a + area_b - intersectionArea;
    
    // 计算iou值
    float iou = intersectionArea / unionArea;
    
    return iou;
}

__device__ float sigmoid(float x)
{
    return 1.0 / (1.0 + expf(-x));
}


__device__ int argmax(const float* data, int size)
{
    int max_idx = 0;
    float max_val = data[0];
    for (int i = 1; i < size; ++i)
    {
        if (data[i] > max_val)
        {
            max_val = data[i];
            max_idx = i;
        }
    }
    return max_idx;
}


// max_size x (x0, y0, x1, y1, x2, y2, x3, y3, x4, y4, 
// t_size, type, conf, color, x0, y0, x1, y1)

__global__ void merge_nms_kernel(float* decode, float* out, int nums_decode, int* nums_out, 
                        float nms_thresh, float merge_thresh)//, int* thresh, int* out_idx)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        //atomicAdd(nums_out, 1);
        int idx[200];
        int picked[20];
        
        for (int i=0; i<nums_decode; i++) 
        {
            idx[i] = i;
        }
        
        for (int i=0; i<nums_decode-1;i++)
            for (int j=0; j<nums_decode-i-1;j++)
                if (decode[idx[j]*18+12] < decode[idx[j+1]*18+12])
                {
                    int temp = idx[j];
                    idx[j] = idx[j+1];
                    idx[j+1] = temp;
                }
        //for (int i=0; i<nums_decode; i++)
        //{
        //    thresh[i] = int(100*decode[idx[i]*18+12]);
        //    out_idx[i] = idx[i];
        //}
        int picked_size = 0;
        //*nums_out = idx[0];
        for (int i=0; i<nums_decode; i++)
        {
            int idxi = idx[i];
            if (decode[idxi*18+12] == 0) continue;
            bool keep = 1;
            //if(idxi==6) *nums_out += 1;
            for (int j=0; j<picked_size; j++)
            {
                int idxj = picked[j];
                float iou = calc_iou(decode+idxi*18+14, decode+idxj*18+14);
                
                if (iou > nms_thresh || isnan(iou))
                {
                    keep = 0;
                    break;
                }
            }
            if (keep)
            {
                picked[picked_size] = idxi;
                picked_size++;
            }
        }

        for (int i=0;i<picked_size;i++)
        {
            int idx_out = idx[picked[i]];
            for (int j=0;j<18;j++)
                out[i*18+j] = decode[idx_out*18+j];
        }
        *nums_out = picked_size;
    }
}

__global__ void decode_outputs_kernel(float *prob, float* l_anchor,
                                      const int stride, const int num_out, const int out_h, const int out_w,
                                      const int img_w, const int img_h, const float bbox_conf_thresh,
                                      const int classes, const int colors, const int camp,
                                      float* objects, float scale, int* nums)
{
    int na = blockIdx.z * blockDim.z + threadIdx.z;
    int h_id = blockIdx.y * blockDim.y + threadIdx.y;
    int w_id = blockIdx.x * blockDim.x + threadIdx.x;
    //*objects = -999;
    //*nums = int(prob[1999]*10000);
    if (na < 3 && h_id < out_h && w_id < out_w)
    {
        // int data_idx = (na * out_h * out_w + h_id * out_w + w_id) * num_out;
        int data_idx = (na * out_h * out_w + h_id * out_w + w_id) * num_out;
        //*nums = na;
        float obj_conf = sigmoid(prob[data_idx + 4]);
        //float obj_conf = prob[data_idx + 4];
        //prob[data_idx] = 100;
        if (obj_conf > bbox_conf_thresh)
        {
            float *color = &prob[data_idx + 15 + classes];
            float *cls = &prob[data_idx + 15];
            float *size = &prob[data_idx + 15 + classes + colors];
            int col_id = argmax(color, colors);
            int cls_id = argmax(cls, classes);

            int t_size = argmax(size, 2);
            
            double final_conf =
                obj_conf * pow(sigmoid(prob[data_idx +15 + cls_id]) *
                                   sigmoid(prob[data_idx +15 + classes + col_id]) *
                                   sigmoid(prob[data_idx +15 + classes + colors + t_size]),
                               1 / 3.);
            if (final_conf > bbox_conf_thresh)
            {
                int n = atomicAdd(nums, 1)-1;
                float x = (sigmoid(prob[data_idx]) * 2.0 - 0.5 + w_id) * stride;
                float y = (sigmoid(prob[data_idx+1]) * 2.0 - 0.5 + h_id) * stride;
                float w = (sigmoid(prob[data_idx+2]) * 2)*(sigmoid(prob[data_idx+2]) * 2) * l_anchor[na * 2];
                float h = (sigmoid(prob[data_idx+3]) * 2)*(sigmoid(prob[data_idx+3]) * 2) * l_anchor[na * 2 + 1];
                for (int p = 0; p < 5; ++p)
                {
                    float px = (prob[data_idx+5 + p * 2] * l_anchor[na * 2] + w_id * stride) / scale;
                    float py = (prob[data_idx+5 + p * 2 + 1] * l_anchor[na * 2 + 1] + h_id * stride) / scale;
                    px = max(min(px, (float)(img_w)), 0.f);
                    py = max(min(py, (float)(img_h)), 0.f);
                    objects[n*18+p*2] = px;
                    objects[n*18+p*2+1] = py;
                }
                float x0 = (x - w * 0.5) / scale;
                float y0 = (y - h * 0.5) / scale;
                float x1 = (x + w * 0.5) / scale;
                float y1 = (y + h * 0.5) / scale;

                x0 = max(min(x0, (float)(img_w)), 0.f);
                y0 = max(min(y0, (float)(img_h)), 0.f);
                x1 = max(min(x1, (float)(img_w)), 0.f);
                y1 = max(min(y1, (float)(img_h)), 0.f);

                objects[n*18+10] = t_size;
                objects[n*18+11] = cls_id;
                objects[n*18+12] = final_conf;
                objects[n*18+13] = col_id;
                objects[n*18+14] = x0;
                objects[n*18+15] = y0;
                objects[n*18+16] = x1;
                objects[n*18+17] = y1;
            }
        
        }
    }
    
}

void decode_outputs_cu(float *prob, float* objects,
                    int stride, int num_out, const int img_w,
                    const int img_h, s_detector_params& param_, int* nums)
{
    float scale = min(param_.w / (img_w * 1.0), param_.h / (img_h * 1.0));
    //cout << "scale:  " << scale << endl;
    float anchor[6];
    float* l_anchor;
    switch (stride)
    {
    case 4:
        for (int k=0;k<6;k++)
            anchor[k] = param_.a1[k];
        break;
    case 8:
        for (int k=0;k<6;k++)
            anchor[k] = param_.a2[k];
        break;
    case 16:
        for (int k=0;k<6;k++)
            anchor[k] = param_.a3[k];
        break;
    case 32:
        for (int k=0;k<6;k++)
            anchor[k] = param_.a4[k];
        break;
    default:
        assert(false && "Unknown layer stride!");
        break;
    }

    int out_h = param_.h / stride;
    int out_w = param_.w / stride;

    dim3 blockSize(32, 32);
    dim3 gridSize((out_h + blockSize.x - 1) / blockSize.x, (out_w + blockSize.y - 1) / blockSize.y, 3);

    hipMalloc(&l_anchor, sizeof(float) * 6);
    hipMemcpy(l_anchor, anchor, sizeof(float)*6, hipMemcpyHostToDevice);

    decode_outputs_kernel<<<gridSize, blockSize>>>(prob, l_anchor,
                                                   stride, num_out, out_h, out_w,
                                                   img_w, img_h, param_.bbox_conf_thresh,
                                                   param_.classes, param_.colors, param_.camp,
                                                   objects, scale, nums);
    hipDeviceSynchronize();
}
void do_nms(float* objects, float* final_out, int* nums, s_detector_params& param_, int n)
{
    dim3 blockSize(1);
    dim3 gridSize(1);
    //int idx_out[n];
    //int* idx_out_ptr;
    //udaMalloc((void**)&idx_out_ptr, sizeof(int)*n);
    //int thresh_out[n];
    //int* thresh_out_ptr;
    //hipMalloc((void**)&thresh_out_ptr, sizeof(int)*n);
    
    //hipDeviceSynchronize();
    merge_nms_kernel<<<gridSize, blockSize>>>(objects, final_out, 
                                n, nums, 
                            param_.nms_thresh, param_.merge_thresh);//,
                            //idx_out_ptr, thresh_out_ptr);
    //hipMemcpy(idx_out, idx_out_ptr, sizeof(int)*n, hipMemcpyDeviceToHost);
    //hipMemcpy(thresh_out, thresh_out_ptr, sizeof(int)*n, hipMemcpyDeviceToHost);
    //for (int i=0;i<n;i++)
    //{
    //    cout << thresh_out[i] << " ";
    //}
    //cout << endl;
    //for (int i=0;i<n;i++)
    //{
    //    cout << idx_out[i] << " ";
    //}
    hipDeviceSynchronize();

}
